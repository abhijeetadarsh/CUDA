#include <hip/hip_runtime.h>
#include <stdio.h>

// macros
#define N (1<<10)
#define BD 256
#define CHECK(call)\
{\
	const hipError_t error = call;\
	if(error != hipSuccess){\
		fprintf(stderr, "Error: %s:%d, ",__FILE__, __LINE__);\
		fprintf(stderr,"code:%d,reason:%s\n",error,\
		hipGetErrorString(error));\
		exit(1);\
	}\
}
#define rec_init float elapsedTime;\
	hipEvent_t start, stop;\
	CHECK(hipEventCreate(&start));\
	CHECK(hipEventCreate(&stop))
#define rec_start CHECK(hipEventRecord(start,0))
#define rec_stop CHECK(hipEventRecord(stop,0));\
	CHECK(hipEventSynchronize(stop));\
	hipEventElapsedTime(&elapsedTime,start,stop)
#define rec_pr(s) printf(s" %3.6f ms\n",elapsedTime)

// functions
void initialize(float *a){
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
	}
}

__global__ void sumReduce(float *dev_a,float *dev_sum)
{
	__shared__ float partialSum[BD];
	partialSum[threadIdx.x] = dev_a[blockIdx.x*blockDim.x + threadIdx.x];
	unsigned int t = threadIdx.x;

	for(unsigned int stride = 1; stride < blockDim.x; stride *= 2)
	{
		__syncthreads();
		if(t % (2*stride) == 0)
		{
			partialSum[t] += partialSum[t+stride];
		}
	}
	if(t == 0) dev_sum[blockIdx.x] = partialSum[0];
}

int main(int argc,char **argv)
{
	// variable declarition
	int bdimx = BD;
	int gdimx = (N + bdimx -1)/bdimx;
	dim3 block(bdimx);
	dim3 grid(gdimx);

	// declaring input array(a) and sum array(b) for both host and device
	float *a,*b;
	float *dev_a,*dev_sum;
	
	a = (float *)malloc(N*sizeof(float));
	b = (float *)malloc(gdimx*sizeof(float));
	// setup for measure time elapsed
	hipEventCreateallocate the memory on device
	CHECK(hipMalloc((void**)&dev_a, N*sizeof(float)));
	CHECK(hipMalloc((void**)&dev_sum, gdimx*sizeof(float)));

	// initilize array a
	initialize(a);

	rec_start;
		// copying array data to device
		CHECK(hipMemcpy(dev_a, a, N*sizeof(float),hipMemcpyHostToDevice));
	rec_stop;

	// printing array size and time elapsed for memory transfer
	printf("Array Size is = %d\n",N);
	rec_pr("Time to do memory transfer of array a, from host to device is");
	
	rec_start;
		//kernel launch
		sumReduce<<<grid,block>>>(dev_a,dev_sum);
		sumReduce<<<1,block>>>(dev_sum,dev_sum);
		hipDeviceSynchronize();
		CHECK(hipMemcpy(b,dev_sum, sizeof(float),hipMemcpyDeviceToHost));
	rec_stop;
	rec_pr("Time to do sum reduction is");
	printf("Sum = %f\n",b[0]);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_sum);
	return 0;
}
