#include <stdio.h>
#include <hip/hip_runtime.h>

// macros:
#define widthField 8
#define precisionField 0

// forward declaration:
class Matrix;
__global__ void init_GPU(double *p, int rows, int cols);
__global__ void mult_GPU(double *d_MatA, double *d_MatB, double *d_MatC, int rows, int x, int cols);

class Matrix
{
public:
	int rows, cols;
	double *device_pointer, *host_pointer;
	// constructor
	Matrix() : rows(0), cols(0), device_pointer(NULL), host_pointer(NULL){};
	Matrix(int r, int c) //: Matrix()
	{
		rows = r;
		cols = c;
		memAllocInBoth();
	}
	Matrix(const Matrix &M)
	{
// #if SHOW_FUNCTION_CALLS == 1
// 		printf("\033[90mMatrix (const Matrix &M)\033[m\n");
// #endif
		rows = M.rows;
		cols = M.cols;
		hipMalloc(&device_pointer, rows * cols * sizeof(double));
		hipMemcpy(device_pointer, M.device_pointer, rows * cols * sizeof(double), hipMemcpyDeviceToDevice);
		host_pointer = (double *)(malloc(rows * cols * sizeof(double)));
		memcpy(host_pointer, M.host_pointer, rows * cols * sizeof(double));
		return;
	}
// 	Matrix(Matrix &&M)
// 	{
// #if SHOW_FUNCTION_CALLS == 1
// 		printf("\033[90mMatrix (Matrix &&M)\033[m\n");
// #endif
// 		rows = M.rows;
// 		cols = M.cols;
// 		device_pointer = M.device_pointer;
// 		host_pointer = M.host_pointer;
// 		M.rows = M.cols = 0;
// 		M.device_pointer = M.host_pointer = NULL;
// 		return;
// 	}
	void memAllocInBoth()
	{
		host_pointer = (double *)malloc(rows * cols * sizeof(double));
		hipMalloc(&device_pointer, rows * cols * sizeof(double));
	}
// 	void display()
// 	{
// 		if (NULL == host_pointer)
// 		{
// #if WARNINGS == 1
// 			printf("\nIn function \'\e[33mprint_matrix_yu\e[m\':\n\e[35mwarning:\e[m \'m\' is (null)\n");
// #endif
// 			return;
// 		}
// #define BUFFER_SIZE 128
// 		int *max_width_arr = (int *)(malloc(cols * sizeof(int)));
// 		char **mat_of_strs = (char **)malloc(rows * cols * sizeof(char *));
// 		char *str;
// 		int width;
// 		for (size_t i = 0; i < cols; i++)
// 		{
// 			max_width_arr[i] = 1;
// 			for (size_t j = 0; j < rows; j++)
// 			{
// 				str = (char *)malloc(BUFFER_SIZE * sizeof(char));
// 				width = snprintf(str, BUFFER_SIZE, "%.*lf", precisionField, host_pointer[j * cols + i]);
// 				str = (char *)realloc(str, ((size_t)(width + 1)) * sizeof(char));
// 				mat_of_strs[j * cols + i] = str;
// 				if (max_width_arr[i] < width)
// 					max_width_arr[i] = width;
// 			}
// 		}
// 		for (size_t i = 0; i < rows; i++)
// 		{
// 			printf("\033[1;32m\xb3\033[m");
// 			for (size_t j = 0; j < cols; j++)
// 			{
// 				width = strlen(mat_of_strs[i * cols + j]);
// 				for (int x = 0; x < max_width_arr[j] - width; x++)
// 					printf(" ");
// 				printf("%s", mat_of_strs[i * cols + j]);
// 				if (j != (cols - 1))
// 					printf(" ");
// 			}
// 			printf("\033[1;32m\xb3\033[m");
// 			printf("\n");
// 		}
// 		for (size_t i = 0; i < rows; i++)
// 			for (size_t j = 0; j < cols; j++)
// 				free(mat_of_strs[i * cols + j]);
// 		free(mat_of_strs);
// 		free(max_width_arr);
// 		return;
// 	}
    void display(){
        for (int i = 0; i < rows; i++)
        {
            for (int j = 0; j < cols; j++)
            {
                printf("%*.*lf", widthField, precisionField, host_pointer[i * cols + j]);
            }
            printf("\n");
        }
    }
	void init()
	{
		dim3 block(1);
		dim3 grid(rows, cols);
		init_GPU<<<grid, block>>>(device_pointer, rows, cols);
		hipDeviceSynchronize();
		D2H();
	}
	void H2D()
	{
		hipMemcpy(device_pointer, host_pointer, rows * cols * sizeof(double), hipMemcpyHostToDevice);
	}
	void D2H()
	{
		hipMemcpy(host_pointer, device_pointer, rows * cols * sizeof(double), hipMemcpyDeviceToHost);
	}
	// operator overload
	Matrix operator*(const Matrix &M)
	{
		if (cols != M.rows)
		{
			printf("Multiplication not valid\n");
			return Matrix();
		}
		Matrix product(rows, M.cols);
		dim3 block(1);
		dim3 grid(rows, M.cols);
		mult_GPU<<<grid, block>>>(device_pointer, M.device_pointer, product.device_pointer, rows, cols, M.cols);
		hipDeviceSynchronize();
		product.D2H();
		return product;
	}
	Matrix operator=(Matrix &M)
	{
		rows = M.rows;
		cols = M.cols;
		memAllocInBoth();
		hipMemcpy(device_pointer, M.device_pointer, rows * cols * sizeof(double), hipMemcpyDeviceToDevice);
		memcpy(host_pointer, M.host_pointer, rows * cols * sizeof(double));
		return *this;
	}
	// Matrix operator=(Matrix &&M)
	// {
	// 	rows = M.rows;
	// 	cols = M.cols;
	// 	device_pointer = M.device_pointer;
	// 	host_pointer = M.host_pointer;
	// 	return *this;
	// }
	// distructor
	~Matrix()
	{
		// printf("\033[31m%p;%p\033[m\n", device_pointer, host_pointer);
		if (device_pointer != NULL)
			hipFree(device_pointer);
		if (host_pointer != NULL)
			free(host_pointer);
	}
};
__global__ void init_GPU(double *p, int rows, int cols)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i < rows && j < cols)
		p[i * cols + j] = (double)(i * cols + j);
	return;
}
__global__ void mult_GPU(double *d_MatA, double *d_MatB, double *d_MatC, int rows, int x, int cols)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i < rows && j < cols)
	{
		double dotp = 0;
		for (int k = 0; k < x; k++)
			dotp += d_MatA[i * x + k] * d_MatB[k * cols + j];
		d_MatC[i * cols + j] = dotp;
	}
	return;
}
int main()
{
	Matrix A(4, 5), B(5, 6), C(6, 7);
	A.init(), B.init(), C.init();
	printf("MATRIX A\n");
	A.display();
	printf("MATRIX B\n");
	B.display();
	printf("MATRIX C\n");
	C.display();
	Matrix ABC = A * B * C;
	printf("MATRIX A x B x C\n");
	ABC.display();
	hipDeviceReset();
	return 0;
}